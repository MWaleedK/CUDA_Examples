#include "hip/hip_runtime.h"
//FFS, WALEED! PLEASE REMEMBER: blocks DO NOT share memory. Threads in a single block DO.
#include<iostream>
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<hip/device_functions.h>
#include ""

int const N = 200;

__global__ void staticMemKernel(int *d_a,int n)
{
	__shared__ int cache[N];
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int tidRev = n - tid - 1;
	
	if (tid < N)
	{
			cache[tid] = d_a[tidRev];
			//printf("(tid: %d,value: %d)\n", tid, cache[tid]);
	}
	__syncthreads();
	
	//printf("(%d,%d)\n", d_a[tid], cache[tidRev]);
	if (tid < N)
	{
		d_a[tid]=cache[tid];
	}


}

__global__ void dynamicMemKernel(int *d_a,int n)
{
	extern __shared__ int cache[];
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int tidRev = n - tid - 1;
	if (tid < N)
	{
		cache[tid] = d_a[tidRev];
		tid += blockDim.x *gridDim.x;
	}
	__syncthreads();

	if (tid < N)
	{
		d_a[tid] = cache[tid];
		
	}
}


void main()
{
	int *h_a = new int[N];
	int *h_b = new int[N];
	int *d_a1;
	int *d_a2;
	for (int i = 0; i < N; i++)
	{
		h_a[i] = i;
	}

	hipMalloc((void**)&d_a1, sizeof(int)*N);
	hipMalloc((void**)&d_a2,sizeof(int)*N);
	
	hipMemcpy(d_a1, h_a,N* sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_a2, h_a,N* sizeof(int), hipMemcpyHostToDevice);

	staticMemKernel << < 1,N>> >(d_a1,N);

	dynamicMemKernel << <1,N,sizeof(int)*N >> >(d_a2, N);
	hipMemcpy(h_a,d_a1,sizeof(int)*N,hipMemcpyDeviceToHost);
	hipMemcpy(h_b, d_a1, sizeof(int)*N, hipMemcpyDeviceToHost);
	for (int count = 0; count < N; count++)
	{
		if (h_a[count] != h_b[count])
		{
			printf("Error\n");
			break;
		}
		else {
			std::cout <<h_a[count]<<std::endl ;
		}
	}
	


	hipFree(d_a2);
	hipFree(d_a1);
	delete[]h_a;
	delete[]h_b;
}